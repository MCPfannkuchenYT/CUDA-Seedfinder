#include "hip/hip_runtime.h"
#include "perlin.hu"

#define AMPLITUDE 1.111111
#define LACUNA 0.333333
#define PERSIST 0.0156250
#define SKIPJRND 1572
#define MULTIP 1.01812688822

__device__ void make_octave(int64_t* seed, perlin_noise* noise) {
    // Generate abc values
    noise->a = nextDouble(seed) * 256.0;
    noise->b = nextDouble(seed) * 256.0;
    noise->c = nextDouble(seed) * 256.0;

    // Fill d with base values
    int i = 0;
    for (i = 0; i < 512; i++) {
        noise->d[i] = 0;
    }
    for (i = 0; i < 256; i++) {
        noise->d[i] = i;
    }

    // Add noise to d
    for (i = 0; i < 256; i++) {
        int bound = 256 - i;
        int m = bound - 1;
        int n3;
        if ((m & bound) == 0) {
            n3 = nextIntPower(seed, 256 - i) + i;
        } else {
            n3 = nextInt(seed, 256 - i) + i;
        }
        int n4 = noise->d[i];
        noise->d[i] = noise->d[n3];
        noise->d[n3] = n4;
        noise->d[i + 256] = noise->d[i];
    }
}

// Makes a nether noise layer
__device__ void make_nether_layer(nether_noise *noise, int64_t structure_seed) {
    int64_t seed;

    // Generate first 2 octaves of each perlin generator of the double perlin generator for temperature
    scramble(&seed, structure_seed);
	skipNextN(&seed, SKIPJRND);
	make_octave(&seed, &noise->temperature.octA.octave0);
	make_octave(&seed, &noise->temperature.octA.octave1);
	skipNextN(&seed, SKIPJRND);
	make_octave(&seed, &noise->temperature.octB.octave0);
	make_octave(&seed, &noise->temperature.octB.octave1);

    // Generate last 2 octaves of each perlin generator of the double perlin generator for humidity
    scramble(&seed, structure_seed+1);
	skipNextN(&seed, SKIPJRND);
	make_octave(&seed, &noise->humidity.octA.octave0);
	make_octave(&seed, &noise->humidity.octA.octave1);
	skipNextN(&seed, SKIPJRND);
	make_octave(&seed, &noise->humidity.octB.octave0);
	make_octave(&seed, &noise->humidity.octB.octave1);
}

__device__ double maintain_precision(double x) {
    return x - floor(x / 33554432.0 + 0.5) * 33554432.0;
}

__device__ double lerp(double part, double from, double to) {
    return from + part * (to - from);
}

__device__ __constant__ double cEdgeX[] = {1.0,-1.0, 1.0,-1.0, 1.0,-1.0, 1.0,-1.0, 0.0, 0.0, 0.0, 0.0,  1.0, 0.0,-1.0, 0.0};
__device__ __constant__ double cEdgeY[] = {1.0, 1.0,-1.0,-1.0, 0.0, 0.0, 0.0, 0.0, 1.0,-1.0, 1.0,-1.0,  1.0,-1.0, 1.0,-1.0};
__device__ __constant__ double cEdgeZ[] = {0.0, 0.0, 0.0, 0.0, 1.0, 1.0,-1.0,-1.0, 1.0, 1.0,-1.0,-1.0,  0.0, 1.0, 0.0,-1.0};

__device__ double indexedLerp(int idx, double d1, double d2, double d3) {
    idx &= 0xf;
    return cEdgeX[idx] * d1 + cEdgeY[idx] * d2 + cEdgeZ[idx] * d3;
}

// Samples a 3D point of a given noise with y 0
__device__ double sample_perlin(perlin_noise *rnd, double d1, double d2, double d3) {
    d1 += rnd->a;
    d2 += rnd->b;
    d3 += rnd->c;
    int i1 = (int)d1 - (int)(d1 < 0);
    int i2 = (int)d2 - (int)(d2 < 0);
    int i3 = (int)d3 - (int)(d3 < 0);
    d1 -= i1;
    d2 -= i2;
    d3 -= i3;
    double t1 = d1*d1*d1 * (d1 * (d1*6.0-15.0) + 10.0);
    double t2 = d2*d2*d2 * (d2 * (d2*6.0-15.0) + 10.0);
    double t3 = d3*d3*d3 * (d3 * (d3*6.0-15.0) + 10.0);

    i1 &= 0xff;
    i2 &= 0xff;
    i3 &= 0xff;

    int a1 = rnd->d[i1]   + i2;
    int a2 = rnd->d[a1]   + i3;
    int a3 = rnd->d[a1+1] + i3;
    int b1 = rnd->d[i1+1] + i2;
    int b2 = rnd->d[b1]   + i3;
    int b3 = rnd->d[b1+1] + i3;

    printf("%f, %f, %f, %d, %d\n", d1, d3, d3, a3, rnd->d[a3]);

    double l1 = indexedLerp(rnd->d[a2],   d1,   d2,   d3);
    double l2 = indexedLerp(rnd->d[b2],   d1-1, d2,   d3);
    double l3 = indexedLerp(rnd->d[a3],   d1,   d2-1, d3);
    double l4 = indexedLerp(rnd->d[b3],   d1-1, d2-1, d3);
    double l5 = indexedLerp(rnd->d[a2+1], d1,   d2,   d3-1);
    double l6 = indexedLerp(rnd->d[b2+1], d1-1, d2,   d3-1);
    double l7 = indexedLerp(rnd->d[a3+1], d1,   d2-1, d3-1);
    double l8 = indexedLerp(rnd->d[b3+1], d1-1, d2-1, d3-1);

    printf("%f, %f, %f, %f, %f, %f, %f, %f\n", l1, l2, l3, l4, l5, l6, l7, l8);

    l1 = lerp(t1, l1, l2);
    l3 = lerp(t1, l3, l4);
    l5 = lerp(t1, l5, l6);
    l7 = lerp(t1, l7, l8);

    l1 = lerp(t2, l1, l3);
    l5 = lerp(t2, l5, l7);

    return lerp(t3, l1, l5);
}

// Samples a 3D point of a given octave with y 0
__device__ double sample_octave(octave_noise *noise, double x, double z) {
    double persist = PERSIST;
    double lacuna = LACUNA;
    double v = 0;
    
    double ax = maintain_precision(x * persist);
    double ay = maintain_precision(0);
    double az = maintain_precision(z * persist);
    printf("%f, %f, %f\n", ax, ay, az);
    v += lacuna * sample_perlin(&noise->octave0, ax, ay, az);
    printf("%f\n", v);
    persist *= 0.5;
    lacuna *= 2.0;

    ax = maintain_precision(x * persist);
    az = maintain_precision(z * persist);
    printf("%f, %f, %f\n", ax, ay, az);
    v += lacuna * sample_perlin(&noise->octave1, ax, ay, az);
    printf("%f\n", v);

    return v;
}

// Samples a 3D point of a given double perlin layer with y 0
__device__ double sample_double_perlin(double_perlin_noise* noise, int32_t x, int32_t z) {
    return (sample_octave(&noise->octA, x, z) + sample_octave(&noise->octB, x*MULTIP, z*MULTIP)) * AMPLITUDE;
}

// Obtains the nether biome at a given position
__device__ int get_nether_biome(NetherNoise *noise, int x, int z) {
    const float npoints[5][4] = {
        { 0,    0,      0,              0       },
        { 0,   -0.5,    0,              1     },
        { 0.4,  0,      0,              2     },
        { 0,    0.5,    0.375*0.375,    3     },
        {-0.5,  0,      0.175*0.175,    4     },
    };

    float temp = sample_double_perlin(&noise->temperature, x, z);
    float humidity = sample_double_perlin(&noise->humidity, x, z);


    printf("%f, %f\n", temp, humidity);

    int i, id = 0;
    float dmin = 0xfffffd00;
    float dmin2 = 0xfffffd00;
    for (i = 0; i < 5; i++) {
        float dx = npoints[i][0] - temp;
        float dy = npoints[i][1] - humidity;
        float dsq = dx*dx + dy*dy + npoints[i][2];
        printf("%f\n", dsq);
        if (dsq < dmin) {
            dmin2 = dmin;
            dmin = dsq;
            id = i;
        } else if (dsq < dmin2) {
            dmin2 = dsq;
        }
    }

    id = (int) npoints[id][3];
    return id;
}