#include "hip/hip_runtime.h"
#include "searcher.hu"

#define A 341873128712L
#define B 132897987541L

__global__ void startSearch(cond* condition, uint32_t conditioncount, uint64_t structureSeedOffset) {
    // Figure out what seed to check
    uint64_t index = ((structureSeedOffset + threadIdx.x) + (blockIdx.x * 1024));
    uint64_t structureSeed = index << 16;

    uint32_t regionX = condition[0].regionX;
    uint32_t regionZ = condition[0].regionZ;
    uint32_t spacing = condition[0].spacing;
    uint32_t offset = condition[0].offset;
    uint64_t salt = condition[0].salt;

    // Find the first structure on that seed
    uint64_t xz = locate_structure(structureSeed,
        // Position Seed Part
        regionX * A + regionZ * B, 
        // Spaced X and Z Region Coordinates
        regionX * spacing, 
        regionZ * spacing, 
        // Salt and Offset
        offset, salt
    );
    uint32_t x = xz >> 32;
    uint32_t z = xz;

    printf("On seed %llu, the first condition was met at %d, %d.\n", structureSeed, x, z);
}